#include "hip/hip_runtime.h"
#include "search_kernel.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define BLOCK_SIZE 256
#define PRIME 101
#define MAX_PATTERN_LENGTH 1024

// Macro for CUDA error checking
#define CHECK_CUDA_CALL(call)                                                  \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: error code %d\n", __FILE__, __LINE__, err); \
      return 0;                                                                \
    }                                                                          \
  } while (0)

__device__ __constant__ char d_pattern[MAX_PATTERN_LENGTH];
__device__ __constant__ unsigned long d_pattern_len;

// Device function to calculate a hash
__device__ unsigned long calculate_hash(const char *str, unsigned long len) {
    unsigned long hash = 0;
    for (unsigned long i = 0; i < len; i++) {
        hash = (hash * PRIME + str[i]);
    }
    return hash;
}

// CUDA kernel for Rabin-Karp algorithm
__global__ void rabin_karp_kernel(const char *text, unsigned long text_len, int *results) {
    const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= text_len - d_pattern_len + 1)
        return;

    unsigned long pattern_hash = calculate_hash(d_pattern, d_pattern_len);
    unsigned long text_hash = calculate_hash(&text[idx], d_pattern_len);

    if (pattern_hash == text_hash) {
        int match = 1;
        for (unsigned long i = 0; i < d_pattern_len; i++) {
            if (text[idx + i] != d_pattern[i]) {
                match = 0;
                break;
            }
        }
        if (match) {
            results[idx] = 1;
        }
    }
}

extern "C" int cuda_rabin_karp_search(const char *pattern, const char *text, unsigned long pattern_len, unsigned long text_len) {
    if (!pattern || !text || pattern_len > MAX_PATTERN_LENGTH)
        return 0;

    char *d_text = NULL;
    int *d_results = NULL;
    int *h_results = NULL;
    int found = 0;

    // Copy pattern to constant memory
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern), pattern, pattern_len));
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_len), &pattern_len, sizeof(unsigned long)));

    // Allocate device memory
    CHECK_CUDA_CALL(hipMalloc(&d_text, text_len));
    CHECK_CUDA_CALL(hipMalloc(&d_results, (text_len - pattern_len + 1) * sizeof(int)));

    // Copy text to device memory
    CHECK_CUDA_CALL(hipMemcpy(d_text, text, text_len, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemset(d_results, 0, (text_len - pattern_len + 1) * sizeof(int)));

    // Launch the kernel
    const unsigned long grid_size = (text_len - pattern_len + BLOCK_SIZE - 1) / BLOCK_SIZE;
    rabin_karp_kernel<<<grid_size, BLOCK_SIZE>>>(d_text, text_len, d_results);

    // Check for kernel launch errors
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Kernel launch error\n");
        hipFree(d_text);
        hipFree(d_results);
        return 0;
    }

    // Copy results back to host memory
    h_results = (int *)malloc((text_len - pattern_len + 1) * sizeof(int));
    if (!h_results) {
        fprintf(stderr, "Host memory allocation failed\n");
        hipFree(d_text);
        hipFree(d_results);
        return 0;
    }

    CHECK_CUDA_CALL(hipMemcpy(h_results, d_results, (text_len - pattern_len + 1) * sizeof(int), hipMemcpyDeviceToHost));

    // Check if any match was found
    for (unsigned long i = 0; i < text_len - pattern_len + 1; i++) {
        if (h_results[i]) {
            found = 1;
            break;
        }
    }

    // Cleanup
    free(h_results);
    hipFree(d_text);
    hipFree(d_results);

    return found;
}

extern "C" bool cuda_batch_search(const char *pattern, char **file_contents, int file_count, size_t *file_sizes) {
    if (!pattern || !file_contents || !file_sizes || file_count <= 0) {
        return false;
    }

    unsigned long pattern_len = strlen(pattern);
    if (pattern_len > MAX_PATTERN_LENGTH) {
        return false;
    }

    // Copy pattern to constant memory
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern), pattern, pattern_len));
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_len), &pattern_len, sizeof(unsigned long)));

    char **d_contents = (char **)malloc(file_count * sizeof(char *));
    int **d_results = (int **)malloc(file_count * sizeof(int *));
    if (!d_contents || !d_results) {
        free(d_contents);
        free(d_results);
        return false;
    }

    bool found_match = false;

    for (int i = 0; i < file_count; i++) {
        size_t text_len = file_sizes[i];
        if (text_len < pattern_len)
            continue;

        CHECK_CUDA_CALL(hipMalloc(&d_contents[i], text_len));
        CHECK_CUDA_CALL(hipMalloc(&d_results[i], (text_len - pattern_len + 1) * sizeof(int)));

        CHECK_CUDA_CALL(hipMemcpy(d_contents[i], file_contents[i], text_len, hipMemcpyHostToDevice));
        CHECK_CUDA_CALL(hipMemset(d_results[i], 0, (text_len - pattern_len + 1) * sizeof(int)));

        const unsigned long grid_size = (text_len - pattern_len + BLOCK_SIZE - 1) / BLOCK_SIZE;
        rabin_karp_kernel<<<grid_size, BLOCK_SIZE>>>(d_contents[i], text_len, d_results[i]);

        if (hipGetLastError() != hipSuccess) {
            fprintf(stderr, "Kernel launch error\n");
            for (int j = 0; j <= i; j++) {
                hipFree(d_contents[j]);
                hipFree(d_results[j]);
            }
            free(d_contents);
            free(d_results);
            return false;
        }
    }

    for (int i = 0; i < file_count; i++) {
        size_t result_size = file_sizes[i] - pattern_len + 1;
        int *h_results = (int *)malloc(result_size * sizeof(int));
        if (!h_results) {
            for (int j = 0; j <= i; j++) {
                hipFree(d_contents[j]);
                hipFree(d_results[j]);
            }
            free(d_contents);
            free(d_results);
            return false;
        }

        CHECK_CUDA_CALL(hipMemcpy(h_results, d_results[i], result_size * sizeof(int), hipMemcpyDeviceToHost));

        for (size_t j = 0; j < result_size; j++) {
            if (h_results[j]) {
                found_match = true;
                break;
            }
        }

        free(h_results);
        hipFree(d_contents[i]);
        hipFree(d_results[i]);

        if (found_match) {
            break;
        }
    }

    free(d_contents);
    free(d_results);

    return found_match;
}
