#include "hip/hip_runtime.h"
#include "search_kernel.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define BLOCK_SIZE 256
#define WARP_SIZE 32
#define PRIME 101
#define MAX_PATTERN_LENGTH 1024

// Macro for CUDA error checking
#define CHECK_CUDA_CALL(call)                                                  \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,       \
              hipGetErrorString(err));                                        \
      return 0;                                                                \
    }                                                                          \
  } while (0)

#define CHECK_CUDA_CALL_BOOL(call)                                             \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,       \
              hipGetErrorString(err));                                        \
      return false;                                                            \
    }                                                                          \
  } while (0)

__device__ __constant__ char d_pattern[MAX_PATTERN_LENGTH];
__device__ __constant__ unsigned long d_pattern_len;
__device__ __constant__ unsigned long d_pattern_hash;

// Precompute pattern hash on host
static unsigned long host_calculate_hash(const char *str, unsigned long len) {
    unsigned long hash = 0;
    for (unsigned long i = 0; i < len; i++) {
        hash = (hash * PRIME + str[i]);
    }
    return hash;
}

// Device function to calculate rolling hash
__device__ unsigned long calculate_hash(const char *str, unsigned long len) {
    unsigned long hash = 0;
    #pragma unroll 8
    for (unsigned long i = 0; i < len; i++) {
        hash = (hash * PRIME + str[i]);
    }
    return hash;
}

// Optimized kernel with warp-level primitives and early exit
__global__ void rabin_karp_kernel(const char *text,
                                  unsigned long text_len,
                                  int *found) {
    const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= text_len - d_pattern_len + 1)
        return;

    // Early exit if already found
    if (*found)
        return;

    // Calculate hash for this position
    unsigned long text_hash = calculate_hash(&text[idx], d_pattern_len);

    // Quick hash comparison first
    if (text_hash == d_pattern_hash) {
        // Verify character-by-character
        bool match = true;
        #pragma unroll 4
        for (unsigned long i = 0; i < d_pattern_len; i++) {
            if (text[idx + i] != d_pattern[i]) {
                match = false;
                break;
            }
        }

        if (match) {
            atomicExch(found, 1);  // Set found flag
        }
    }
}

__global__ void batch_rabin_karp_kernel(char **texts,
                                        unsigned long *text_lens,
                                        int file_count,
                                        int *found) {
    const int file_idx = blockIdx.y;
    const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (file_idx >= file_count)
        return;

    if (*found)
        return;

    const char *text = texts[file_idx];
    const unsigned long text_len = text_lens[file_idx];

    if (idx >= text_len - d_pattern_len + 1)
        return;

    unsigned long text_hash = calculate_hash(&text[idx], d_pattern_len);

    if (text_hash == d_pattern_hash) {
        bool match = true;
        #pragma unroll 4
        for (unsigned long i = 0; i < d_pattern_len; i++) {
            if (text[idx + i] != d_pattern[i]) {
                match = false;
                break;
            }
        }

        if (match) {
            atomicExch(found, 1);
        }
    }
}

extern "C" int cuda_rabin_karp_search(const char *pattern,
                                      const char *text,
                                      unsigned long pattern_len,
                                      unsigned long text_len) {
    if (!pattern || !text || pattern_len > MAX_PATTERN_LENGTH || pattern_len > text_len)
        return 0;

    // Precompute pattern hash on host
    unsigned long pattern_hash = host_calculate_hash(pattern, pattern_len);

    char *d_text = NULL;
    int *d_found = NULL;
    int h_found = 0;

    // Copy pattern and metadata to constant memory
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern),       // symbol
                                       pattern,          // src
                                       pattern_len));    // count
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_len),   // symbol
                                       &pattern_len,     // src
                                       sizeof(unsigned long)));  // count
    CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_hash),  // symbol
                                       &pattern_hash,    // src
                                       sizeof(unsigned long)));  // count

    // Allocate device memory
    CHECK_CUDA_CALL(hipMalloc(&d_text, text_len));             // devPtr, size
    CHECK_CUDA_CALL(hipMalloc(&d_found, sizeof(int)));         // devPtr, size

    // Copy data to device
    CHECK_CUDA_CALL(hipMemcpy(d_text,                          // dst
                               text,                            // src
                               text_len,                        // count
                               hipMemcpyHostToDevice));        // kind
    CHECK_CUDA_CALL(hipMemset(d_found, 0, sizeof(int)));       // devPtr, value, count

    // Launch kernel with optimal grid size
    const unsigned long num_positions = text_len - pattern_len + 1;
    const unsigned long grid_size = (num_positions + BLOCK_SIZE - 1) / BLOCK_SIZE;

    rabin_karp_kernel<<<grid_size, BLOCK_SIZE>>>(d_text,       // text
                                                  text_len,     // text_len
                                                  d_found);     // found

    // Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
        hipFree(d_text);
        hipFree(d_found);
        return 0;
    }

    // Copy result back
    CHECK_CUDA_CALL(hipMemcpy(&h_found,                        // dst
                               d_found,                         // src
                               sizeof(int),                     // count
                               hipMemcpyDeviceToHost));        // kind

    // Cleanup
    hipFree(d_text);
    hipFree(d_found);

    return h_found;
}

extern "C" bool cuda_batch_search(const char *pattern,
                                  char **file_contents,
                                  int file_count,
                                  size_t *file_sizes) {
    if (!pattern || !file_contents || !file_sizes || file_count <= 0)
        return false;

    unsigned long pattern_len = strlen(pattern);
    if (pattern_len > MAX_PATTERN_LENGTH)
        return false;

    // Precompute pattern hash
    unsigned long pattern_hash = host_calculate_hash(pattern, pattern_len);

    // Copy pattern to constant memory
    CHECK_CUDA_CALL_BOOL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern),         // symbol
                                            pattern,            // src
                                            pattern_len));      // count
    CHECK_CUDA_CALL_BOOL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_len),     // symbol
                                            &pattern_len,       // src
                                            sizeof(unsigned long)));  // count
    CHECK_CUDA_CALL_BOOL(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_hash),    // symbol
                                            &pattern_hash,      // src
                                            sizeof(unsigned long)));  // count

    // Allocate unified arrays for all files
    char **d_file_ptrs = NULL;
    unsigned long *d_file_lens = NULL;
    int *d_found = NULL;

    CHECK_CUDA_CALL_BOOL(hipMalloc(&d_file_ptrs,               // devPtr
                                    file_count * sizeof(char*))); // size
    CHECK_CUDA_CALL_BOOL(hipMalloc(&d_file_lens,               // devPtr
                                    file_count * sizeof(unsigned long))); // size
    CHECK_CUDA_CALL_BOOL(hipMalloc(&d_found, sizeof(int)));    // devPtr, size
    CHECK_CUDA_CALL_BOOL(hipMemset(d_found, 0, sizeof(int)));  // devPtr, value, count

    // Allocate device memory for each file and copy data
    char **h_d_file_ptrs = (char **)malloc(file_count * sizeof(char*));
    unsigned long *h_file_lens = (unsigned long *)malloc(file_count * sizeof(unsigned long));

    if (!h_d_file_ptrs || !h_file_lens) {
        free(h_d_file_ptrs);
        free(h_file_lens);
        hipFree(d_file_ptrs);
        hipFree(d_file_lens);
        hipFree(d_found);
        return false;
    }

    // Single batch allocation and copy
    for (int i = 0; i < file_count; i++) {
        h_file_lens[i] = file_sizes[i];

        if (file_sizes[i] < pattern_len) {
            h_d_file_ptrs[i] = NULL;
            continue;
        }

        CHECK_CUDA_CALL_BOOL(hipMalloc(&h_d_file_ptrs[i],              // devPtr
                                        file_sizes[i]));                 // size
        CHECK_CUDA_CALL_BOOL(hipMemcpy(h_d_file_ptrs[i],               // dst
                                        file_contents[i],                // src
                                        file_sizes[i],                   // count
                                        hipMemcpyHostToDevice));        // kind
    }

    // Copy pointer arrays to device
    CHECK_CUDA_CALL_BOOL(hipMemcpy(d_file_ptrs,                        // dst
                                    h_d_file_ptrs,                       // src
                                    file_count * sizeof(char*),          // count
                                    hipMemcpyHostToDevice));            // kind
    CHECK_CUDA_CALL_BOOL(hipMemcpy(d_file_lens,                        // dst
                                    h_file_lens,                         // src
                                    file_count * sizeof(unsigned long),  // count
                                    hipMemcpyHostToDevice));            // kind

    // Find max file size for grid sizing
    size_t max_len = 0;
    for (int i = 0; i < file_count; i++) {
        if (file_sizes[i] > max_len)
            max_len = file_sizes[i];
    }

    // Launch batch kernel with 2D grid
    const unsigned long max_positions = max_len - pattern_len + 1;
    dim3 grid((max_positions + BLOCK_SIZE - 1) / BLOCK_SIZE,    // x
              file_count);                                       // y
    dim3 block(BLOCK_SIZE);                                      // x

    batch_rabin_karp_kernel<<<grid, block>>>(d_file_ptrs,       // texts
                                             d_file_lens,        // text_lens
                                             file_count,         // file_count
                                             d_found);           // found

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
        for (int i = 0; i < file_count; i++) {
            if (h_d_file_ptrs[i])
                hipFree(h_d_file_ptrs[i]);
        }
        free(h_d_file_ptrs);
        free(h_file_lens);
        hipFree(d_file_ptrs);
        hipFree(d_file_lens);
        hipFree(d_found);
        return false;
    }

    // Copy result
    int h_found = 0;
    CHECK_CUDA_CALL_BOOL(hipMemcpy(&h_found,                           // dst
                                    d_found,                            // src
                                    sizeof(int),                        // count
                                    hipMemcpyDeviceToHost));           // kind

    // Cleanup
    for (int i = 0; i < file_count; i++) {
        if (h_d_file_ptrs[i])
            hipFree(h_d_file_ptrs[i]);
    }
    free(h_d_file_ptrs);
    free(h_file_lens);
    hipFree(d_file_ptrs);
    hipFree(d_file_lens);
    hipFree(d_found);

    return h_found != 0;
}
